#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

#include <aliceVision/depthMap/cuda/deviceCommon/device_matrix.cu>
#include <aliceVision/depthMap/cuda/images/gauss_filter.hpp>

namespace aliceVision {
namespace depthMap {

#ifdef TSIM_USE_FLOAT
using TSim = float;
using TSimAcc = float;
#else
using TSim = unsigned char;
using TSimAcc = unsigned int; // TSimAcc is the similarity accumulation type
#endif

using TSimRefine = float;

inline __device__ void volume_computePatch( int rc_cam_cache_idx,
                                            int tc_cam_cache_idx,
                                            Patch& ptch,
                                            const float fpPlaneDepth, const int2& pix )
{
    ptch.p = get3DPointForPixelAndFrontoParellePlaneRC(rc_cam_cache_idx, pix, fpPlaneDepth); // no texture use
    ptch.d = computePixSize(rc_cam_cache_idx, ptch.p); // no texture use
    computeRotCSEpip(rc_cam_cache_idx, tc_cam_cache_idx, ptch); // no texture use
}

__global__ void volume_init_kernel(TSim* volume, int volume_s, int volume_p,
                                    int volDimX, int volDimY )
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z; // * blockDim.z + threadIdx.z;

    if(vx >= volDimX || vy >= volDimY)
        return;

    *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz) = 255.0f;
}

__global__ void volume_init_kernel(TSimRefine* volume, int volume_s, int volume_p, 
                                   int volDimX, int volDimY, TSimRefine value)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z; // * blockDim.z + threadIdx.z;

    if(vx >= volDimX || vy >= volDimY)
        return;

    *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz) = value;
}

__global__ void volume_add_kernel(TSimRefine* out_volume, int out_volume_s, int out_volume_p,
                                  const TSimRefine* volume, int volume_s, int volume_p,
                                  int volDimX, int volDimY )
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z;

    if(vx >= volDimX || vy >= volDimY)
        return;

    TSimRefine* outSim = get3DBufferAt(out_volume, out_volume_s, out_volume_p, vx, vy, vz);
    *outSim += *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz); 
    // *outSim += min(*outSim, *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz));
}

__global__ void volume_initFromSimMap_kernel(TSimRefine* volume, int volume_s, int volume_p, 
                                             float* simMap, int simMap_p, 
                                             int zIndex, int volDimX, int volDimY)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z;

    if(vx >= volDimX || vy >= volDimY)
        return;

//    if(vz == zIndex)
//    {
//        // const float fsim = *get2DBufferAt(simMap, simMap_p, vx, vy) * 255.0f;
//        *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz) = TSimRefine(255.0f); // TODO: simMap not used, here simMap is pixSize map
//    }
//    else
//    {
        *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz) = TSimRefine(255.0f);
//    }
}

__global__ void volume_slice_kernel(
                                    hipTextureObject_t rc_tex,
                                    hipTextureObject_t tc_tex,
                                    int rc_cam_cache_idx,
                                    int tc_cam_cache_idx,
                                    const float* depths_d,
                                    const int startDepthIndex,
                                    const int nbDepthsToSearch,
                                    int rcWidth, int rcHeight,
                                    int tcWidth, int tcHeight,
                                    int wsh,
                                    const float gammaC, const float gammaP,
                                    TSim* volume_1st, int volume1st_s, int volume1st_p,
                                    TSim* volume_2nd, int volume2nd_s, int volume2nd_p,
                                    int volStepXY,
                                    int volDimX, int volDimY)
{
    /*
     * Note !
     * volDimX == width  / volStepXY
     * volDimY == height / volStepXY
     * width and height are needed to compute transformations,
     * volDimX and volDimY may be the number of samples, reducing memory or computation
     */

    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z; // * blockDim.z + threadIdx.z;

    if( vx >= volDimX || vy >= volDimY ) // || vz >= volDimZ
        return;
    // if (vz >= nbDepthsToSearch)
    //  return;
    assert(vz < nbDepthsToSearch);

    const int x = vx * volStepXY;
    const int y = vy * volStepXY;

    // if(x >= rcWidth || y >= rcHeight)
    //     return;

    const int zIndex = startDepthIndex + vz;
    const float fpPlaneDepth = depths_d[zIndex];

    Patch ptcho;
    volume_computePatch( rc_cam_cache_idx,
                         tc_cam_cache_idx,
                         ptcho, fpPlaneDepth, make_int2(x, y)); // no texture use

    float fsim = compNCCby3DptsYK(rc_tex, tc_tex,
                                  rc_cam_cache_idx, tc_cam_cache_idx,
                                  ptcho, wsh,
                                  rcWidth, rcHeight,
                                  tcWidth, tcHeight,
                                  gammaC, gammaP);

    constexpr const float fminVal = -1.0f;
    constexpr const float fmaxVal = 1.0f;
    constexpr const float fmultiplier = 1.0f / (fmaxVal - fminVal);

    if(fsim == HIP_INF_F) // invalid similarity
    {
      fsim = 255.0f;
    }
    else // valid similarity
    {
      fsim = (fsim - fminVal) * fmultiplier;

#ifdef TSIM_USE_FLOAT
      // no clamp
#else
      fsim = fminf(1.0f, fmaxf(0.0f, fsim));
#endif
      // convert from (0, 1) to (0, 254)
      // needed to store in the volume in uchar
      // 255 is reserved for the similarity initialization, i.e. undefined values
      fsim *= 254.0f;
    }

    TSim* fsim_1st = get3DBufferAt(volume_1st, volume1st_s, volume1st_p, vx, vy, zIndex);
    TSim* fsim_2nd = get3DBufferAt(volume_2nd, volume2nd_s, volume2nd_p, vx, vy, zIndex);

    if (fsim < *fsim_1st)
    {
        *fsim_2nd = *fsim_1st;
        *fsim_1st = TSim(fsim);
    }
    else if (fsim < *fsim_2nd)
    {
        *fsim_2nd = TSim(fsim);
    }
}

__global__ void volume_refine_kernel(hipTextureObject_t rc_tex, 
                                     hipTextureObject_t tc_tex, 
                                     int rcamCacheId,
                                     int tcamCacheId, 
                                     int rcWidth, int rcHeight, 
                                     int tcWidth, int tcHeight,
                                     int wsh, float gammaC, float gammaP, 
                                     float* depthMap_d, int depthMap_p, 
                                     TSimRefine* volume_d, int volume_s, int volume_p, 
                                     int volStepXY, int volDimX, int volDimY, int volDimZ)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z; // relative depth index

    if(vx >= volDimX || vy >= volDimY)
        return;

    const int x = vx * volStepXY;
    const int y = vy * volStepXY;
    const float originalDepth = *get2DBufferAt(depthMap_d, depthMap_p, x, y); // input original depth

    // original depth invalid or masked, similarity value remain at 255
    if(originalDepth < 0.0f) 
        return; 

    // get rc 3d point at original depth (z center)
    float3 p = get3DPointForPixelAndDepthFromRC(rcamCacheId, make_int2(x, y), originalDepth);

    // move rc 3d point according to the relative depth
    const int relativeDepthIndexOffset = vz - ((volDimZ - 1) / 2);
    if(relativeDepthIndexOffset != 0)
    {
        const float pixSizeOffset = relativeDepthIndexOffset * computePixSize(rcamCacheId, p);
        move3DPointByRcPixSize(rcamCacheId, p, pixSizeOffset);
    }

    // compute patch
    Patch ptch;
    ptch.p = p;
    ptch.d = computePixSize(rcamCacheId, p);
    computeRotCSEpip(rcamCacheId, tcamCacheId, ptch);

    // compute similarity
    float fsim = compNCCby3DptsYK(rc_tex, tc_tex, 
                                  rcamCacheId, 
                                  tcamCacheId, 
                                  ptch, wsh, 
                                  rcWidth, rcHeight,
                                  tcWidth, tcHeight, 
                                  gammaC, gammaP);


    constexpr const float fminVal = -1.0f;
    constexpr const float fmaxVal = 1.0f;
    constexpr const float fmultiplier = 1.0f / (fmaxVal - fminVal);

    if(fsim == HIP_INF_F) // invalid similarity
    {
        fsim = 255.0f;
    }
    else // valid similarity
    {
        //fsim = (fsim - fminVal) * fmultiplier;

//#ifdef TSIM_USE_FLOAT
        // no clamp
//#else
        //fsim = fminf(1.0f, fmaxf(0.0f, fsim));
//#endif
        // convert from (0, 1) to (0, 254)
        // needed to store in the volume in uchar
        // 255 is reserved for the similarity initialization, i.e. undefined values
        //fsim *= 254.0f;
    }

    const float fsimInvertedFiltered = sigmoid(0.0f, 1.0f, 0.7f, -0.7f, fsim);

    TSimRefine* outSim = get3DBufferAt(volume_d, volume_s, volume_p, vx, vy, vz);

    if(fsim < *outSim)
    {
        *outSim = TSimRefine(fsimInvertedFiltered);
    }
}

__global__ void volume_gauss_smooth_z_kernel(TSimRefine* out_volume_d, int out_volume_s, int out_volume_p, 
                                             const TSimRefine* volume_d, int volume_s, int volume_p, 
                                             int volDimX, int volDimY, int volDimZ, int radius)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z;

    const int gaussScale = radius - 1;

    if(vx >= volDimX || vy >= volDimY)
        return;

    float sum = 0.0f;
    float sumFactor = 0.0f;

    for(int rz = -radius; rz <= radius; rz++)
    {
        const int iz = vz + rz;
        if((iz < volDimZ) && (iz > 0))
        {
            const float value = float(*get3DBufferAt(volume_d, volume_s, volume_p, vx, vy, iz));
            const float factor = getGauss(gaussScale, rz + radius);
            sum += value * factor;
            sumFactor += factor;
        }
    }

    *get3DBufferAt(out_volume_d, out_volume_s, out_volume_p, vx, vy, vz) = TSimRefine(sum / sumFactor);
}

__global__ void volume_gauss_smooth_xyz_kernel(TSimRefine* out_volume_d, int out_volume_s, int out_volume_p,
                                               const TSimRefine* volume_d, int volume_s, int volume_p,
                                               int volDimX, int volDimY, int volDimZ, int radius)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z;

    const int gaussScale = radius - 1;

    if(vx >= volDimX || vy >= volDimY)
        return;

    const int xMinRadius = max(-radius, -vx);
    const int yMinRadius = max(-radius, -vy);
    const int zMinRadius = max(-radius, -vz);

    const int xMaxRadius = min(radius, volDimX - vx - 1);
    const int yMaxRadius = min(radius, volDimY - vy - 1);
    const int zMaxRadius = min(radius, volDimZ - vz - 1);

    float sum = 0.0f;
    float sumFactor = 0.0f;

    for(int rx = xMinRadius; rx <= xMaxRadius; rx++)
    {
        const int ix = vx + rx;

        for(int ry = yMinRadius; ry <= yMaxRadius; ry++)
        {
            const int iy = vy + ry;

            for(int rz = zMinRadius; rz <= zMaxRadius; rz++)
            {
                const int iz = vz + rz;
   
                const float value = float(*get3DBufferAt(volume_d, volume_s, volume_p, ix, iy, iz));
                const float factor = getGauss(gaussScale, rx + radius) * getGauss(gaussScale, ry + radius) * getGauss(gaussScale, rz + radius);
                sum += value * factor;
                sumFactor += factor;
            }
        }
    }

    *get3DBufferAt(out_volume_d, out_volume_s, out_volume_p, vx, vy, vz) = TSimRefine(sum / sumFactor);
}

__device__ float depthPlaneToDepth(
    int cam_cache_idx,
    const float2& pix,
    float fpPlaneDepth)
{
    const CameraStructBase& cam = camsBasesDev[cam_cache_idx];
    float3 planen = M3x3mulV3(cam.iR, make_float3(0.0f, 0.0f, 1.0f));
    normalize(planen);
    float3 planep = cam.C + planen * fpPlaneDepth;
    float3 v = M3x3mulV2(cam.iP, pix);
    normalize(v);
    float3 p = linePlaneIntersect(cam.C, v, planep, planen);
    float depth = size(cam.C - p);
    return depth;
}


__global__ void volume_retrieveBestZ_kernel(
  int rcamCacheId,
  float* bestDepthM, int bestDepthM_s,
  float* bestSimM, int bestSimM_s,
  const TSim* simVolume, int simVolume_s, int simVolume_p,
  int volDimX, int volDimY, int volDimZ,
  const float* depths_d,
  int scaleStep, bool interpolate)
{
  const int x = blockIdx.x * blockDim.x + threadIdx.x;
  const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if(x >= volDimX || y >= volDimY)
    return;

  float bestSim = 255.0f;
  int bestZIdx = -1;
  for (int z = 0; z < volDimZ; ++z)
  {
    const float simAtZ = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, z);
    if (simAtZ < bestSim)
    {
      bestSim = simAtZ;
      bestZIdx = z;
    }
  }

  // TODO: consider filtering out the values with a too bad score like (bestSim > 200.0f)
  //       to reduce the storage volume of the depth maps
  if (bestZIdx == -1)
  {
      *get2DBufferAt(bestDepthM, bestDepthM_s, x, y) = -1.0f;
      *get2DBufferAt(bestSimM, bestSimM_s, x, y) = 1.0f;
      return;
  }

  const float2 pix{float(x * scaleStep), float(y * scaleStep)};
  // Without depth interpolation (for debug purpose only)
  if(!interpolate)
  {
    *get2DBufferAt(bestDepthM, bestDepthM_s, x, y) = depthPlaneToDepth(rcamCacheId, pix, depths_d[bestZIdx]);
    *get2DBufferAt(bestSimM, bestSimM_s, x, y) = (bestSim / 255.0f) * 2.0f - 1.0f; // convert from (0, 255) to (-1, +1)
    return;
  }

  // With depth/sim interpolation
  const int bestZIdx_m1 = max(0, bestZIdx - 1);
  const int bestZIdx_p1 = min(volDimZ-1, bestZIdx + 1);

  float3 depths;
  depths.x = depths_d[bestZIdx_m1];
  depths.y = depths_d[bestZIdx];
  depths.z = depths_d[bestZIdx_p1];

  float3 sims;
  sims.x = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, bestZIdx_m1);
  sims.y = bestSim;
  sims.z = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, bestZIdx_p1);

  // Convert sims from (0, 255) to (-1, +1)
  sims.x = (sims.x / 255.0f) * 2.0f - 1.0f;
  sims.y = (sims.y / 255.0f) * 2.0f - 1.0f;
  sims.z = (sims.z / 255.0f) * 2.0f - 1.0f;

  // Interpolation between the 3 depth planes candidates
  const float refinedDepth = refineDepthSubPixel(depths, sims);

  *get2DBufferAt(bestDepthM, bestDepthM_s, x, y) = depthPlaneToDepth(rcamCacheId, pix, refinedDepth);
  *get2DBufferAt(bestSimM, bestSimM_s, x, y) = sims.y;
}

__global__ void volume_refineFuseBestZ_kernel(int rcamCacheId, 
                                              float* bestDepthMap_d, int bestDepthMap_p,
                                              float* bestSimMap_d, int bestSimMap_p, 
                                              const float* originalDepthMap_d, int originalDepthMap_p, 
                                              const TSimRefine* simVolume, int simVolume_s, int simVolume_p, 
                                              int volDimX, int volDimY, int volDimZ, int volScaleStepXY,
                                              float samplesPerPixSize, float twoTimesSigmaPowerTwo, float nbSamplesHalf,
                                              bool interpolate)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;

    const int x = vx * volScaleStepXY;
    const int y = vy * volScaleStepXY;

    if(vx >= volDimX || vy >= volDimY)
        return;

    const float originalDepth = *get2DBufferAt(originalDepthMap_d, originalDepthMap_p, x, y); // input original depth

    if(originalDepth < 0.0f) // original depth invalid or masked
    {
        *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = originalDepth; // -1 (invalid) or -2 (masked)
        *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = 1.0f;              // similarity between (-1, +1)
        return;
    }

    // find best z sample per pixel
    float bestSampleSim = 99999.f;
    int bestSampleOffsetIndex = 0;

    for(int s = -nbSamplesHalf; s <= nbSamplesHalf; ++s)
    {
        float sampleSim = 0.0f;

        for(int vz = 0; vz < volDimZ; ++vz)
        {
            const int rz = (vz - ((volDimZ - 1) / 2)); // depth relative index offset
            const int zs = rz * samplesPerPixSize;     // depth relative samples offset

            float fsim = (float(*get3DBufferAt(simVolume, simVolume_s, simVolume_p, vx, vy, vz)) / 255.f) * 2 - 1; // converted from (0,255) to (-1,1)

            if(interpolate) // for now, average
            {
                int nbNeighbors = 0;
                const int vz_m1 = vz - 1;
                const int vz_p1 = vz + 1;

                if(vz_m1 >= 0)
                {
                    fsim += (float(*get3DBufferAt(simVolume, simVolume_s, simVolume_p, vx, vy, vz_m1)) / 255.f) * 2 - 1; // converted from (0,255) to (-1,1)
                    ++nbNeighbors;
                }

                if(vz_p1 < volDimZ)
                {
                    fsim += (float(*get3DBufferAt(simVolume, simVolume_s, simVolume_p, vx, vy, vz_p1)) / 255.f) * 2 - 1; // converted from (0,255) to (-1,1)
                    ++nbNeighbors;
                }
                
                fsim = fsim / (1 + nbNeighbors);

            }

            const float fsimFiltered = -sigmoid(0.0f, 1.0f, 0.7f, -0.7f, fsim);

            sampleSim += fsimFiltered * expf(-((zs - s) * (zs - s)) / twoTimesSigmaPowerTwo);
        }

        if(sampleSim < bestSampleSim)
        {
            bestSampleSim = sampleSim;
            bestSampleOffsetIndex = s;
        }
    }

    // get rc 3d point at original depth (z center)
    const float3 p = get3DPointForPixelAndDepthFromRC(rcamCacheId, make_int2(x, y), originalDepth);
    const float sampleSize = computePixSize(rcamCacheId, p) / samplesPerPixSize;
    const float sampleSizeOffset = bestSampleOffsetIndex * sampleSize;
    const float bestDepth = originalDepth + sampleSizeOffset;

    // without depth interpolation (for debug purpose only)
    //if(!interpolate)
    {
        *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = bestDepth;
        *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = bestSampleSim;
        return;
    }
}


__global__ void volume_refineBestZ_kernel(int rcamCacheId, 
                                          float* bestDepthMap_d, int bestDepthMap_p, 
                                          float* bestSimMap_d, int bestSimMap_p, 
                                          const float* originalDepthMap_d, int originalDepthMap_p, 
                                          const TSimRefine* simVolume, int simVolume_s, int simVolume_p,
                                          int volDimX, int volDimY, int volDimZ, 
                                          int volStepXY, bool interpolate)
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;

    const int x = vx * volStepXY;
    const int y = vy * volStepXY;

    if(vx >= volDimX || vy >= volDimY)
        return;

    const float originalDepth = *get2DBufferAt(originalDepthMap_d, originalDepthMap_p, x, y); // input original depth

    if(originalDepth < 0.0f) // original depth invalid or masked
    {
        *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = originalDepth; // -1 (invalid) or -2 (masked)
        *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = 1.0f; // similarity between (-1, +1)
        return;
    }

    float bestSim = 255.0f;
    int bestZIdx = -1;

    for(int z = 0; z < volDimZ; ++z)
    {
        const float simAtZ = float(*get3DBufferAt(simVolume, simVolume_s, simVolume_p, vx, vy, z));
        if(simAtZ < bestSim)
        {
            bestSim = simAtZ;
            bestZIdx = z;
        }
    }

    if(bestZIdx == -1)
    {
        *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = -1.0f; // invalid depth
        *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = 1.0f; // similarity between (-1, +1)
        return;
    }

    // get rc 3d point at original depth (z center)
    float3 p = get3DPointForPixelAndDepthFromRC(rcamCacheId, make_int2(x, y), originalDepth);

    const int relativeDepthIndexOffset = bestZIdx - ((volDimZ - 1) / 2);
    const float pixSize = computePixSize(rcamCacheId, p);
    const float pixSizeOffset = relativeDepthIndexOffset * pixSize;
    const float bestDepth = originalDepth + pixSizeOffset;


    // without depth interpolation (for debug purpose only)
    if(!interpolate)
    {
        *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = bestDepth;
        *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = (bestSim / 255.0f) * 2.0f - 1.0f; // convert from (0, 255) to (-1, +1)
        return;
    }

    // with depth interpolation
    const int bestZIdx_m1 = max(0, bestZIdx - 1);
    const int bestZIdx_p1 = min(volDimZ - 1, bestZIdx + 1);
    const float pixSizeOffset_m1 = (bestZIdx_m1 - ((volDimZ - 1) / 2)) * pixSize; // relative depth index offset m1 * pixSize
    const float pixSizeOffset_p1 = (bestZIdx_p1 - ((volDimZ - 1) / 2)) * pixSize; // relative depth index offset p1 * pixSize

    float3 depths;
    depths.x = originalDepth + pixSizeOffset_m1;
    depths.y = bestDepth;
    depths.z = originalDepth + pixSizeOffset_p1;

    float3 sims;
    sims.x = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, bestZIdx_m1);
    sims.y = bestSim;
    sims.z = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, bestZIdx_p1);

    // convert sims from (0, 255) to (-1, +1)
    sims.x = (sims.x / 255.0f) * 2.0f - 1.0f;
    sims.y = (sims.y / 255.0f) * 2.0f - 1.0f;
    sims.z = (sims.z / 255.0f) * 2.0f - 1.0f;

    // interpolation between the 3 depth candidates
    *get2DBufferAt(bestDepthMap_d, bestDepthMap_p, x, y) = refineDepthSubPixel(depths, sims);
    *get2DBufferAt(bestSimMap_d, bestSimMap_p, x, y) = sims.y;

}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__global__ void volume_initVolumeYSlice_kernel(T* volume, int volume_s, int volume_p, const int3 volDim, const int3 axisT, int y, T cst)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;

    int3 v;
    (&v.x)[axisT.x] = x;
    (&v.x)[axisT.y] = y;
    (&v.x)[axisT.z] = z;

    if ((x >= 0) && (x < (&volDim.x)[axisT.x]) && (z >= 0) && (z < (&volDim.x)[axisT.z]))
    {
        T* volume_zyx = get3DBufferAt(volume, volume_s, volume_p, v.x, v.y, v.z);
        *volume_zyx = cst;
    }
}

template <typename T1, typename T2>
__global__ void volume_getVolumeXZSlice_kernel(T1* slice, int slice_p,
                                               const T2* volume, int volume_s, int volume_p,
                                               const int3 volDim, const int3 axisT, int y)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;

    int3 v;
    (&v.x)[axisT.x] = x;
    (&v.x)[axisT.y] = y;
    (&v.x)[axisT.z] = z;

    if (x >= (&volDim.x)[axisT.x] || z >= (&volDim.x)[axisT.z])
      return;

    const T2* volume_xyz = get3DBufferAt(volume, volume_s, volume_p, v);
    T1* slice_xz = get2DBufferAt(slice, slice_p, x, z);
    *slice_xz = (T1)(*volume_xyz);
}

__global__ void volume_computeBestZInSlice_kernel(TSimAcc* xzSlice, int xzSlice_p, TSimAcc* ySliceBestInColCst, int volDimX, int volDimZ)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    if(x >= volDimX)
        return;

    TSimAcc bestCst = *get2DBufferAt(xzSlice, xzSlice_p, x, 0);

    for(int z = 1; z < volDimZ; ++z)
    {
        const TSimAcc cst = *get2DBufferAt(xzSlice, xzSlice_p, x, z);
        bestCst = cst < bestCst ? cst : bestCst;  // min(cst, bestCst);
    }
    ySliceBestInColCst[x] = bestCst;
}

/**
 * @param[inout] xySliceForZ input similarity plane
 * @param[in] xySliceForZM1
 * @param[in] xSliceBestInColCst
 * @param[out] volSimT output similarity volume
 */
__global__ void volume_agregateCostVolumeAtXinSlices_kernel(
            hipTextureObject_t rc_tex,
            TSimAcc* xzSliceForY, int xzSliceForY_p,
            const TSimAcc* xzSliceForYm1, int xzSliceForYm1_p,
            const TSimAcc* bestSimInYm1,
            TSim* volAgr, int volAgr_s, int volAgr_p,
            const int3 volDim,
            const int3 axisT,
            float step,
            int y, float _P1, float _P2,
            int ySign, int filteringIndex)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.y * blockDim.y + threadIdx.y;

    int3 v;
    (&v.x)[axisT.x] = x;
    (&v.x)[axisT.y] = y;
    (&v.x)[axisT.z] = z;

    if (x >= (&volDim.x)[axisT.x] || z >= volDim.z)
        return;

    TSimAcc* sim_xz = get2DBufferAt(xzSliceForY, xzSliceForY_p, x, z);
    float pathCost = 255.0f;

    if((z >= 1) && (z < volDim.z - 1))
    {
        float P2 = 0;

        if(_P2 < 0)
        {
          // _P2 convention: use negative value to skip the use of deltaC.
          P2 = std::abs(_P2);
        }
        else
        {
          const int imX0 = v.x * step; // current
          const int imY0 = v.y * step;

          const int imX1 = imX0 - ySign * step * (axisT.y == 0); // M1
          const int imY1 = imY0 - ySign * step * (axisT.y == 1);

          const float4 gcr0 = tex2D_float4(rc_tex, float(imX0) + 0.5f, float(imY0) + 0.5f);
          const float4 gcr1 = tex2D_float4(rc_tex, float(imX1) + 0.5f, float(imY1) + 0.5f);
          const float deltaC = Euclidean3(gcr0, gcr1);

          // sigmoid f(x) = i + (a - i) * (1 / ( 1 + e^(10 * (x - P2) / w)))
          // see: https://www.desmos.com/calculator/1qvampwbyx
          // best values found from tests: i = 80, a = 255, w = 80, P2 = 100
          // historical values: i = 15, a = 255, w = 80, P2 = 20
          P2 = sigmoid(80.f, 255.f, 80.f, _P2, deltaC);
        }

        const TSimAcc bestCostInColM1 = bestSimInYm1[x];
        const TSimAcc pathCostMDM1 = *get2DBufferAt(xzSliceForYm1, xzSliceForYm1_p, x, z - 1); // M1: minus 1 over depths
        const TSimAcc pathCostMD   = *get2DBufferAt(xzSliceForYm1, xzSliceForYm1_p, x, z);
        const TSimAcc pathCostMDP1 = *get2DBufferAt(xzSliceForYm1, xzSliceForYm1_p, x, z + 1); // P1: plus 1 over depths
        const float minCost = multi_fminf(pathCostMD, pathCostMDM1 + _P1, pathCostMDP1 + _P1, bestCostInColM1 + P2);

        // if 'pathCostMD' is the minimal value of the depth
        pathCost = (*sim_xz) + minCost - bestCostInColM1;
    }

    // fill the current slice with the new similarity score
    *sim_xz = TSimAcc(pathCost);

#ifndef TSIM_USE_FLOAT
    // clamp if TSim = uchar (TSimAcc = unsigned int)
    pathCost = fminf(255.0f, fmaxf(0.0f, pathCost));
#endif

    // aggregate into the final output
    TSim* volume_xyz = get3DBufferAt(volAgr, volAgr_s, volAgr_p, v.x, v.y, v.z);
    const float val = (float(*volume_xyz) * float(filteringIndex) + pathCost) / float(filteringIndex + 1);
    *volume_xyz = TSim(val);
}

} // namespace depthMap
} // namespace aliceVision
